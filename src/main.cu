#include "hip/hip_runtime.h"

#include <GLFW/glfw3.h>
#include <GL/gl.h>
#include <stdio.h>
#include <vector>
#include <iostream>
#include <sstream> 
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <cmath>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <iomanip>
#include <string>
#include <csignal>

int width = 1024;
int height = 1024;

int GRID_X, GRID_Y;
bool double_dim;

int block_size;

std::string method;

std::string CUDA = "CUDA";
std::string CPU = "CPU";

std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
std::chrono::high_resolution_clock::time_point stop = std::chrono::high_resolution_clock::now();

std::chrono::high_resolution_clock::time_point frame_start = std::chrono::high_resolution_clock::now();
std::chrono::high_resolution_clock::time_point frame_end = std::chrono::high_resolution_clock::now(); 

bool running = true;

int num_elements, grid_size;

std::vector<char> cells, next_cells;
std::vector<std::vector<char>> cells_double, next_cells_double;

char* d_cells, *d_next_cells, *d_cells_double, *d_next_cells_double;

hipError_t err;

float zoom_factor = 1.0f;

void signal_handler(int signal) {
    if (signal == SIGINT) {
        std::cout << "\n Stopping program..." << std::endl;
        running = false;
    }
}

void print_config(){

    using namespace std;

    cout << "Grid Columns: " << GRID_X << endl;
    cout << "Grid Rows: " << GRID_Y << endl;
    cout << "Method: " << method << endl;
    
    if (method == CUDA){
        cout << "Double dimension: " << (double_dim ? "true" : "false") << endl;
        cout << "Block size: " << block_size << endl;
    }

}

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods) {
 
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
        running = false;
        std::cout << "\nESC key pressed. Exiting loop." << std::endl;
    }
}

void init_vectors() {
     
    cells = std::vector<char>(GRID_X * GRID_Y, 0);
    next_cells = std::vector<char>(GRID_X * GRID_Y, 0);

    if (method == CPU) {
        
        // Random number generator for initializing cells
        std::random_device rd;
        std::mt19937 gen(rd());  // Mersenne Twister random number generator
        std::uniform_int_distribution<> dis(0, 1);  // Random values between 0 and 2 (inclusive)

        for (int i = 0; i < GRID_X * GRID_Y; ++i) {
            int rand_value = dis(gen);  // Random value in {0, 1, 2}
            cells[i] = (rand_value < 0.1);  // If rand_value < 3, set cell to true (about 33% chance)
        }

        std::cout << "CPU: Initialized cells and next_cells with random values." << std::endl;
    }
}


int get_neighbours(int x, int y) {
    x = (x + GRID_X) % GRID_X;
    y = (y + GRID_Y) % GRID_Y;

    return cells[(x + 1) % GRID_X + y * GRID_X] +
           cells[(x - 1 + GRID_X) % GRID_X + y * GRID_X] +
           cells[x + ((y + 1) % GRID_Y) * GRID_X] +
           cells[x + ((y - 1 + GRID_Y) % GRID_Y) * GRID_X] +
           cells[(x + 1) % GRID_X + ((y + 1) % GRID_Y) * GRID_X] +
           cells[(x - 1 + GRID_X) % GRID_X + ((y + 1) % GRID_Y) * GRID_X] +
           cells[(x + 1) % GRID_X + ((y - 1 + GRID_Y) % GRID_Y) * GRID_X] +
           cells[(x - 1 + GRID_X) % GRID_X + ((y - 1 + GRID_Y) % GRID_Y) * GRID_X];
}

void game_of_life_cpu() {
    for (int j = 0; j < GRID_Y; j++) {
        for (int i = 0; i < GRID_X; i++) {
            int neighbors = get_neighbours(i, j);
            int idx = j * GRID_X + i;

            if (cells[idx]) {
                next_cells[idx] = (neighbors == 2 || neighbors == 3);
            } else {
                next_cells[idx] = (neighbors == 3);
            }
        }
    }
    std::swap(cells, next_cells);
}

void draw_grid(int rows, int cols) {

    glBegin(GL_POINTS);

    for (int j = 0; j < rows; j++) {
        for (int i = 0; i < cols; i++) {
            
                float x = i - cols / 2;
                float y = -j + rows / 2;

                if (cells[i + j * cols] == 1) {
                    glColor3f(1.0f, 0.0f, 0.0f);
                } else {
                    glColor3f(0.0f, 0.0f, 0.0f);
                }

                glVertex2f(x, y);
            
        }
    }

    glEnd();

}

void initialize_camera(int rows, int cols, int window_width, int window_height) {

    float zoom_factor = static_cast<float>(std::max(GRID_X*1.0/window_width, GRID_Y*1.0/window_height));

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    float zoomed_width = window_width * zoom_factor;
    float zoomed_height = window_height * zoom_factor;

    glOrtho(-zoomed_width / 2, zoomed_width / 2, -zoomed_height / 2, zoomed_height / 2, -1.0, 1.0);

    glMatrixMode(GL_MODELVIEW);
}

int setup_cuda_memory(int grid_x, int grid_y){

    err = hipMalloc((void**)&d_cells, GRID_X * GRID_Y * sizeof(char));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_cells: " << hipGetErrorString(err) << std::endl;
        return -9;
    }

    err = hipMalloc((void**)&d_next_cells, GRID_X * GRID_Y * sizeof(char));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for d_next_cells: " << hipGetErrorString(err) << std::endl;
        return -10;
    }

    return 0;
}

GLFWwindow* init_glfw(int width, int height){

    if (!glfwInit()) {
        return NULL;
    }

    GLFWwindow* window = glfwCreateWindow(width, height, "Game Of Life", NULL, NULL);

    if (!window) {
        glfwTerminate();
        return NULL;
    }

    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
    glfwSetKeyCallback(window, key_callback);
    glfwMakeContextCurrent(window);
    glViewport(0, 0, width, height);
    glfwSwapInterval(1);
    glfwSetWindowAttrib(window, GLFW_RESIZABLE, GLFW_FALSE);


    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glLineWidth(1.0f);

    glClear(GL_COLOR_BUFFER_BIT); 

    return window;
}

__global__ void game_of_life_kernel(char* d_cells, char* d_next_cells, int grid_x, int grid_y) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < grid_x * grid_y) {

        int x = idx % grid_x;
        int y = idx / grid_x;

        int neighbors = 0;

        for (int dy = -1; dy <= 1; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {
                if (dx == 0 && dy == 0) continue;

                int nx = (x + dx + grid_x) % grid_x;
                int ny = (y + dy + grid_y) % grid_y;

                neighbors += d_cells[nx + ny * grid_x];
            }
        }

        if (d_cells[idx]) {
            d_next_cells[idx] = (neighbors == 2 || neighbors == 3);
        } else {
            d_next_cells[idx] = (neighbors == 3);
        }
    }
}

__global__ void game_of_life_kernel_2d(char* d_cells, char* d_next_cells, int grid_x, int grid_y) {

    int block_x = blockIdx.x * blockDim.x + threadIdx.x;
    int block_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (block_x < grid_x && block_y < grid_y) {

        int neighbors = 0;

        for (int dy = -1; dy <= 1; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {

                if (dx == 0 && dy == 0) continue;

                int nx = (block_x + dx + grid_x) % grid_x;
                int ny = (block_y + dy + grid_y) % grid_y;

                neighbors += d_cells[nx + ny * grid_x];
            }
        }

        int idx = block_x + block_y * grid_x;
        if (d_cells[idx]) {
            d_next_cells[idx] = (neighbors == 2 || neighbors == 3);
        } else {
            d_next_cells[idx] = (neighbors == 3);
        }
    }
}

__global__ void randomize_grid_cuda_kernel(char* d_cells, int grid_x, int grid_y, unsigned long long seed) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < grid_x * grid_y) {
     
        int x = idx % grid_x;
        int y = idx / grid_x;

        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        float random_value = hiprand_uniform(&state);

        if (random_value < 0.1f) {
            d_cells[idx] = 1;
        } else {
            d_cells[idx] = 0;
        }
    }
}


int main(int argc, char** argv) {

    std::signal(SIGINT, signal_handler);

    if(!(argc == 7)){
        return -2;
    }

    int draw = std::atoi(argv[1]);

    GRID_X = std::atoi(argv[2]);

    if (GRID_X <= 0){
        return -3;
    }

    GRID_Y = std::atoi(argv[3]);

    if (GRID_Y <= 0){
        return -4;
    }

    method = argv[4];

 

    double_dim = std::atoi(argv[5]);
    block_size = std::atoi(argv[6]);

    init_vectors();

    dim3 block_size_2d(2, block_size);
    dim3 grid_size_2d((GRID_X + block_size_2d.x - 1) / block_size_2d.x,
                  (GRID_Y + block_size_2d.y - 1) / block_size_2d.y);

    if(method == CUDA){
        num_elements = GRID_X * GRID_Y;
        grid_size = (num_elements + block_size - 1) / block_size;


        if(setup_cuda_memory(GRID_X, GRID_Y)){
            return -11;
        }

        hipMemcpy(d_cells, cells.data(), GRID_X * GRID_Y * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_next_cells, next_cells.data(), GRID_X * GRID_Y * sizeof(char), hipMemcpyHostToDevice);

        randomize_grid_cuda_kernel<<<grid_size, block_size>>>(d_cells,GRID_X, GRID_Y, 1);
    }

    print_config();

    unsigned long long cells_proccesed = 0;

    std::chrono::high_resolution_clock::time_point app_start = std::chrono::high_resolution_clock::now();

    std::chrono::high_resolution_clock::time_point app_end;

    if(draw){
        if (!glfwInit()) {
            fprintf(stderr, "Failed to initialize GLFW\n");
            return -1;
        }

        GLFWwindow* window = init_glfw(width, height);

        if (!window) {
            return -8;
        } 

        initialize_camera(GRID_X,GRID_Y,width,height); 

        while (!glfwWindowShouldClose(window) && running) {

            frame_start = std::chrono::high_resolution_clock::now();

            if (method == CPU) {
        

                game_of_life_cpu();


            } else {

                start = std::chrono::high_resolution_clock::now();

                if(double_dim){
                    game_of_life_kernel_2d<<<grid_size_2d, block_size_2d>>>(d_cells, d_next_cells, GRID_X, GRID_Y);
                } else {
                    game_of_life_kernel<<<grid_size, block_size>>>(d_cells, d_next_cells, GRID_X, GRID_Y);
                }

                hipDeviceSynchronize();
        
                err = hipGetLastError();
                if (err != hipSuccess) {
                    std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
                    break;
                }

                hipMemcpy(cells.data(), d_next_cells, GRID_X * GRID_Y * sizeof(char), hipMemcpyDeviceToHost);
                hipMemcpy(next_cells.data(), d_cells, GRID_X * GRID_Y * sizeof(char), hipMemcpyDeviceToHost);

                stop = std::chrono::high_resolution_clock::now();
                auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        
                char* temp = d_cells;
                d_cells = d_next_cells;
                d_next_cells = temp;

                        
                stop = std::chrono::high_resolution_clock::now();
                duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    
            }
        
            draw_grid(GRID_X, GRID_Y);

            glfwSwapBuffers(window);
            glfwPollEvents();

            frame_end = std::chrono::high_resolution_clock::now();

            app_end = frame_end;
            cells_proccesed+= GRID_X*GRID_Y; 

            auto time_between_frames = std::chrono::duration_cast<std::chrono::microseconds>(frame_end - frame_start);

            auto time_app = std::chrono::duration_cast<std::chrono::microseconds>(app_end - app_start);

            double cells_per_second = cells_proccesed/(time_app.count()/1000000.f);

            int count = time_between_frames.count();

            if (count == 0){
                count = 1000000;
            }


            int fps = 1000000/count;

            std::ostringstream titleStream;
            titleStream << "GameOfLife - FPS: " << fps << " Cells per second: " << cells_per_second;

            glfwSetWindowTitle(window, titleStream.str().c_str());
        }

        glfwDestroyWindow(window);
        glfwTerminate();
    } else { 

        while(running){

        if (method == CPU) {
        
                game_of_life_cpu();
                app_end = std::chrono::high_resolution_clock::now();

            } else {


                if(double_dim){
                    game_of_life_kernel_2d<<<grid_size_2d, block_size_2d>>>(d_cells, d_next_cells, GRID_X, GRID_Y);
                } else {
                    game_of_life_kernel<<<grid_size, block_size>>>(d_cells, d_next_cells, GRID_X, GRID_Y);
                }

                err = hipGetLastError();
                if (err != hipSuccess) {
                    std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
                    return;
                }

                hipDeviceSynchronize();

                hipMemcpy(cells.data(), d_next_cells, GRID_X * GRID_Y * sizeof(char), hipMemcpyDeviceToHost);
                app_end = std::chrono::high_resolution_clock::now();

                char* temp = d_cells;
                d_cells = d_next_cells;
                d_next_cells = temp;
            }
        
            cells_proccesed+=GRID_X*GRID_Y; 

            auto time_app = std::chrono::duration_cast<std::chrono::microseconds>(app_end - app_start);

            double cells_per_second = cells_proccesed/(time_app.count()/1000000.f);

            std::ostringstream titleStream;
    
            std::cout << "\rCells per second: " << static_cast<unsigned long long>(cells_per_second) << std::flush;

        }
    }

    std::cout << "\nMain loop ended" << std::endl;


    if(method == CUDA){
        hipFree(d_cells);
        hipFree(d_next_cells);

        std::cout << "Freed CUDA resources" << std::endl;
    }    

    return 0;
}
